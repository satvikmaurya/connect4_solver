#include "hip/hip_runtime.h"
/**
 * @defgroup   CUDA_SOLVER
 *
 * @brief      This file implements tournament APIs which can allow humans/solvers
 * to play against each other.
 *
 * @author     Robert Viramontes (except where otherwise noted)
 * @date       2021
 */

#include "cudaSolver.cuh"

#include <iostream>
#include "hipblas.h"

// There are 4 streak directions: horizontal, vertical, '/' and '\'
#define NUM_STREAK_DIR 4

// Current implementation seems to be limited to a depth of 6 because of memory requirements
#define MAX_DEPTH 6

// Error check macro
// Based on https://github.com/NVIDIA-developer-blog/code-samples/blob/master/posts/tensor-cores/simpleTensorCoreGEMM.cu
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
inline void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
      abort();
   }
}
__global__ void findSlotsOnBoard(const SlotStatus* concat_board, const int concat_board_len, const SlotStatus color, uint8_t updateBoard[])
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < concat_board_len) {
        updateBoard[idx] = concat_board[idx] == color ? 1 : 0;
    }
}

__global__ void findStreaksForLength(const uint8_t *board, int width, int height, double *scores, int stencilLength, const int startBoard, const int entriesPerBoard) {
    auto numSlots = width*height;
    if (threadIdx.x >= numSlots*NUM_STREAK_DIR)
        return;
    
    int matched = 0;

    bool stencilCanGoRight = threadIdx.x % width <= width - stencilLength;
    bool stencilCanGoLeft = threadIdx.x % width >= stencilLength - 1;
    bool stencilCanGoUp = (threadIdx.x/width ) % height <= height - stencilLength;

    int boardStartIndex = (blockIdx.x + startBoard) * numSlots;

    if (threadIdx.x < numSlots) {
        // Check horizontal scores
        if (stencilCanGoRight) {
            for (int i = 0; i < stencilLength; i++) {
                matched += board[threadIdx.x + i + boardStartIndex];
            }
        }           
    } else if (threadIdx.x < 2*numSlots) {
        // Check vertical scores
        if (stencilCanGoUp) {
            for (int i = 0; i < stencilLength; i++) {
                matched += board[(threadIdx.x-numSlots) + (i*width) + boardStartIndex];
            }
        }
    } else if (threadIdx.x < 3*numSlots) {
        // Check \ diagonal
        if (stencilCanGoLeft && stencilCanGoUp) {
            for (int i = 0; i < stencilLength; i++) {
                matched += board[(threadIdx.x-2*numSlots) + i*width - i + boardStartIndex];
            }
        }
    } else if (threadIdx.x < 4*numSlots) {
        // Check / diagonal
        if (stencilCanGoRight && stencilCanGoUp) {
            for (int i = 0; i < stencilLength; i++) {
                matched += board[(threadIdx.x-3*numSlots) + i*width + i + boardStartIndex];
            }
        }
    }

    scores[threadIdx.x + (stencilLength-2)*numSlots*NUM_STREAK_DIR + (blockIdx.x + startBoard) * entriesPerBoard] = matched == stencilLength ? 1 : 0;
}

CudaSolver::CudaSolver(int width, int height, int winningStreakSize)
{
    _board = new Board(width, height, winningStreakSize);
    // Set a reasonable number of streams so we don't spend a ton of time spinning up streams.
    _numStreams = 32;
    _streams = new hipStream_t[_numStreams];
    hipStream_t* streams = (hipStream_t*)_streams;
    
    for (int i = 0; i < _numStreams; i++) {
        hipStreamCreate(&streams[i]);
    }
}

CudaSolver::~CudaSolver() 
{
    hipStream_t* streams = (hipStream_t*)_streams;
    for (int i = 0; i < _numStreams; i++) {
        hipStreamDestroy(streams[i]);
    } 

    delete[] streams;
}

int CudaSolver::solve(Player player, int maxDepth, double time_limit) 
{
    if (_board->DetermineWinner() != Player::None) {
        return -1;
    }

	_nodesTraversed = 0;
	int retval = -1;
	int bestMove = -1;
	auto nodesTraversed = _nodesTraversed;

	if(time_limit > 0) {
		// Implement Iterative Deepening to adhere to a time limit per move
		this->startTimer();
		for (int depth = 2; this->isTimeLeft(time_limit) && depth <= MAX_DEPTH; depth += 2) {
			// Find the best move
			int move = this->findBestMove2(_board->getBoard(), player, depth);
			if(this->isTimeLeft(time_limit)) {
				bestMove = move;
				nodesTraversed = _nodesTraversed;
				_nodesTraversed = 0;
			}
		}
		_nodesTraversed = nodesTraversed;
	}
	else {
		bestMove = this->findBestMove2(_board->getBoard(), player, maxDepth);
	}

	if(_board->IsFull()) {
		std::cout << "Board is full" << std::endl;
		return -1;
	}

	if(bestMove > -1) {
		_board->playMove(bestMove, SlotStatusHelpers::getSlotFromPlayer(player));
		retval = bestMove % _board->getWidth();
	}

	Player winner = _board->DetermineWinner();
	if(winner != Player::None) {
		std::cout << "Player: " << ((winner == Player::Red)?"Red ":"Yellow ")
		<< "is the winner" << std::endl;
		return -1;
	}
	_totalNodesTraversed += _nodesTraversed;
    return retval;
}

int CudaSolver::EvaluateBoard(SlotStatus* board, Player player)
{
    int numSlots =  _board->getWidth() * _board->getHeight();
    // subtract one from winning streak size since there are no streaks of size 1
    int numStreakLengths = _board->getWinningStreakSize() - 1; 
    int entriesPerBoard = numSlots*NUM_STREAK_DIR*numStreakLengths;
    uint8_t *player_board;
    uint8_t *opp_board;
    hipMallocManaged(&player_board, numSlots*sizeof(uint8_t));
    hipMallocManaged(&opp_board, numSlots*sizeof(uint8_t));

    double* d_player_streak_counts;
    double* d_opp_streak_counts;
    hipMalloc(&d_player_streak_counts, entriesPerBoard*sizeof(double));
    hipMalloc(&d_opp_streak_counts, entriesPerBoard*sizeof(double));

    double *h_x = new double[entriesPerBoard];
    double *h_y = new double[1];
    for (int strk_len = 0; strk_len < numStreakLengths; strk_len++) {
        for (int i = 0; i < numSlots*NUM_STREAK_DIR; i++) {
            h_x[strk_len*numSlots*NUM_STREAK_DIR + i] = pow(strk_len+2, 3);
        }
    }
    h_y[0] = 0;

    double *d_x;
    double *d_y;
    hipMalloc(&d_x, entriesPerBoard*sizeof(double));
    hipMalloc(&d_y, sizeof(double));

    hipMemcpy(d_x, h_x, entriesPerBoard*sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_y, 0, sizeof(double));

    SlotStatus slotToMatch = SlotStatusHelpers::getSlotFromPlayer(player);
    SlotStatus oppSlot = SlotStatusHelpers::getSlotFromPlayer(PlayerHelpers::OppositePlayer(player));
    for (int i = 0; i < numSlots; i++) {
        player_board[i] = board[i] == slotToMatch ? 1 : 0;
        opp_board[i] = board[i] == oppSlot ? 1 : 0;
    }

    hipStream_t player_stream, opp_stream;
    hipStreamCreate(&player_stream);
    hipStreamCreate(&opp_stream);

    for (int n = 0; n < numStreakLengths; n++)
    {
        auto index = n*numSlots*NUM_STREAK_DIR;
        // launch one worker kernel per stream
        findStreaksForLength<<<1,256,0, player_stream>>>(
            player_board, 
            _board->getWidth(), 
            _board->getHeight(), 
            &d_player_streak_counts[index], 
            n+2, 0, entriesPerBoard);
        
        findStreaksForLength<<<1,256,0, opp_stream>>>(
            opp_board, 
            _board->getWidth(), 
            _board->getHeight(), 
            &d_opp_streak_counts[index], 
            n+2, 0, entriesPerBoard);
    }

    // Make sure all kernels are done executing
    hipDeviceSynchronize();

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    double alpha = 1;
    double beta = 0;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    stat = hipblasDgemv(
        handle, HIPBLAS_OP_T,
        entriesPerBoard, 1, 
        &alpha,
        d_player_streak_counts, entriesPerBoard,
        d_x, 1,
        &beta, 
        d_y, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS gemv failed\n");
        return 0;
    }

    alpha = -1;
    beta = 1;

    // auto h_temp = new double[numBoards];
    // hipMemcpy(h_temp, d_y, numBoards*sizeof(double), hipMemcpyDeviceToHost);
    // for (int i = 0; i < numBoards; i++) {
    //     // see if anything happened
    //     std:: cout << h_temp[i] << " ";
    // }
    // std::cout << std::endl;

    stat = hipblasDgemv(
        handle, HIPBLAS_OP_T,
        entriesPerBoard, 1, 
        &alpha,
        d_opp_streak_counts, entriesPerBoard,
        d_x, 1,
        &beta, 
        d_y, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS gemv failed\n");
        return 0;
    }

    hipMemcpy(h_y, d_y, sizeof(double), hipMemcpyDeviceToHost);
    auto score = h_y[0];

    // for (int i = 0; i < num_streams; i++) {
    //     // for (int j = 0; j < 3; j++)
    //     // {
    //     //     std::cout << streak_counts[i][j] << " streaks of " << j+2 << " ";
    //     // }
    //     // std::cout << std::endl;
    //     hipStreamDestroy(streams[i]);
    // }

    hipblasDestroy(handle);
    hipStreamDestroy(player_stream);
    hipStreamDestroy(opp_stream);
    hipFree(player_board);
    hipFree(opp_board);
    hipFree(d_player_streak_counts);
    hipFree(d_opp_streak_counts);
    hipFree(d_x);
    hipFree(d_y);

    delete[] h_x;
    delete[] h_y;

    return score;
}

int CudaSolver::EvaluateBoards(const SlotStatus* concat_boards, const int numBoards, const Player player, double scores[])
{
    hipStream_t* streams = (hipStream_t*)_streams;
    const int boardWidth = _board->getWidth();
    const int boardHeight = _board->getHeight();
    const int numSlots = boardWidth * boardHeight;
    // subtract one from winning streak size since there are no streaks of size 1
    const int numStreakLengths = _board->getWinningStreakSize() - 1; 
    const int concat_boards_len = numSlots*numBoards;
    const int entriesPerBoard = numStreakLengths*numSlots*NUM_STREAK_DIR;

    uint8_t *d_player_board;
    uint8_t *d_opp_board;
    cudaErrCheck( hipMalloc(&d_player_board, concat_boards_len*sizeof(uint8_t)) );
    cudaErrCheck( hipMalloc(&d_opp_board, concat_boards_len*sizeof(uint8_t)) );

    SlotStatus* d_concat_boards;
    cudaErrCheck( hipMalloc(&d_concat_boards, concat_boards_len*sizeof(SlotStatus)) );
    cudaErrCheck( hipMemcpy(d_concat_boards, concat_boards, concat_boards_len*sizeof(SlotStatus), hipMemcpyHostToDevice) );

    SlotStatus slotToMatch = SlotStatusHelpers::getSlotFromPlayer(player);
    SlotStatus oppSlot = SlotStatusHelpers::getSlotFromPlayer(PlayerHelpers::OppositePlayer(player));
    int numThreads = 256;
    int numBlocks = (concat_boards_len/256) + 1;
    findSlotsOnBoard<<<numBlocks, numThreads, 0, streams[0]>>>(d_concat_boards, concat_boards_len, slotToMatch, d_player_board);
    findSlotsOnBoard<<<numBlocks, numThreads, 0, streams[1]>>>(d_concat_boards, concat_boards_len, oppSlot, d_opp_board);

    double* d_player_streak_counts;
    double* d_opp_streak_counts;
    cudaErrCheck( hipMalloc(&d_player_streak_counts, numBoards*entriesPerBoard*sizeof(double)) );
    cudaErrCheck( hipMalloc(&d_opp_streak_counts, numBoards*entriesPerBoard*sizeof(double)) );

    double *h_x;
    cudaErrCheck( hipHostMalloc((void **)&h_x, entriesPerBoard*sizeof(double)) );
    for (int strk_len = 0; strk_len < numStreakLengths; strk_len++) {
        for (int i = 0; i < numSlots*NUM_STREAK_DIR; i++) {
            h_x[strk_len*numSlots*NUM_STREAK_DIR + i] = pow(strk_len+2, 3);
        }
    }

    double *d_x;
    double *d_y;
    cudaErrCheck( hipMalloc(&d_x, entriesPerBoard*sizeof(double)) );
    cudaErrCheck( hipMalloc(&d_y, numBoards*sizeof(double)) );

    // Throw this on the last-used stream so it can queue up and we can start launching kernels without waiting for this, since it's used later
    hipMemcpyAsync(d_x, h_x, entriesPerBoard*sizeof(double), hipMemcpyHostToDevice, streams[_numStreams-1]);
    cudaErrCheck( hipMemsetAsync(d_y, 0, numBoards*sizeof(double), streams[_numStreams-1]) );

    const int boardsPerLaunch = 32;
    int stream = -1;
    for (int b = 0; b < numBoards; b += 32)
    {
        stream++;
        if (stream % _numStreams == 0) { stream = 0; }
        for (int n = 0; n < numStreakLengths; n++)
        {
            int blocksToLaunch = (b + boardsPerLaunch < numBoards) ? boardsPerLaunch : (numBoards % boardsPerLaunch) ;

            // launch one worker kernel per stream
            findStreaksForLength<<<blocksToLaunch,256, 0, streams[stream]>>>(
                d_player_board, 
                boardWidth, 
                boardHeight, 
                d_player_streak_counts,
                n+2,
                b,
                entriesPerBoard);

            findStreaksForLength<<<blocksToLaunch,256, 0, streams[stream]>>>(
                d_opp_board, 
                boardWidth, 
                boardHeight, 
                d_opp_streak_counts,
                n+2,
                b,
                entriesPerBoard);
        }
    }

    hipHostFree(h_x);
    hipFree(d_concat_boards);
    hipFree(d_player_board);
    hipFree(d_opp_board);

    // At this point, we want to make sure that all the work we've queued up is done before we do the matrix multiplies
    // I think this is redundant since hipFree is synchronizing, but keep it just for clarity
    hipDeviceSynchronize();

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    double alpha = 1;
    double beta = 0;

    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return EXIT_FAILURE;
    }

    stat = hipblasDgemv(
        handle, HIPBLAS_OP_T,
        entriesPerBoard, numBoards, 
        &alpha,
        d_player_streak_counts, entriesPerBoard,
        d_x, 1,
        &beta, 
        d_y, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS gemv failed\n");
        return 2;
    }

    // Setting alpha = -1 and beta = 1 allows us to use this call to 
    // do player - opponent (since it's beta*player + alpha*opponent)
    alpha = -1;
    beta = 1;

    stat = hipblasDgemv(
        handle, HIPBLAS_OP_T,
        entriesPerBoard, numBoards, 
        &alpha,
        d_opp_streak_counts, entriesPerBoard,
        d_x, 1,
        &beta, 
        d_y, 1);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf("CUBLAS gemv failed\n");
        return 3;
    }

    hipblasDestroy(handle);

    hipFree(d_player_streak_counts);
    hipFree(d_opp_streak_counts);

    hipMemcpy(scores, d_y, numBoards*sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);

    return 0;
}

bool CudaSolver::isLegalMove(const SlotStatus *board, int width, int height, int index) 
{
    if (board[index] != SlotStatus::Empty) return false;
    int row = index / width;
    if(row == height - 1) return true;
    if(board[index + width] == SlotStatus::Empty) return false;
    return true;
}

void CudaSolver::findBoards(SlotStatus* board, const int width, const int height, const SlotStatus pieceToPlace, SlotStatus* &concatBoard, int &numBoards)
{
    std::vector<SlotStatus *> boards;
    auto numSlots = width*height;
    for (int i = numSlots - 1; i >= 0; i--) {
        if (isLegalMove(board, width, height, i)) {
            auto copied_board = new SlotStatus[width*height];

            // Copy the board over
            for (int j = 0; j < numSlots; j++) copied_board[j] = board[j];

            copied_board[i] = pieceToPlace;
            boards.push_back(copied_board);
        }
    }

    concatBoard = new SlotStatus[boards.size()*numSlots];
    numBoards = boards.size();

    for (int i = 0; i < boards.size(); i++) {
        for(int j = 0; j < numSlots; j++) concatBoard[i*numSlots + j] = boards.at(i)[j];
        delete[] boards.at(i);
    }
}

void CudaSolver::findBoards2(const SlotStatus* board, const int width, const int height, const Player player, const int depth, std::vector<CudaSolver::boardAndPath> *completed_boards, std::vector<int> pathSoFar)
{
    // Iterate through all possible moves to find legal moves.
    for (int i = width * height - 1; i >= 0; i--) {
        if (isLegalMove(board, width, height, i)) {
            auto copied_board = new SlotStatus[width*height];
            std::vector<int> nextPath;
            std::copy(pathSoFar.begin(), pathSoFar.end(), back_inserter(nextPath));
            nextPath.push_back(i);

            // Copy the board over
            for (int j = 0; j < width*height; j++) copied_board[j] = board[j];

            copied_board[i] = SlotStatusHelpers::getSlotFromPlayer(player);

            if (depth == 1) {
                // If we're at the bottom of the stack, add the boards
                // with its path to the completed_boards
                CudaSolver::boardAndPath t;
                t.board = copied_board;
                t.path = nextPath;
                completed_boards->push_back(t);
            } else {
                findBoards2(copied_board, width, height, PlayerHelpers::OppositePlayer(player), depth-1, completed_boards, nextPath);
                delete[] copied_board;
            }
        }
    }
}

uint32_t CudaSolver::createPathMapping(const std::vector<int> *path)
{
    uint32_t pathMapping = 0;
    uint32_t multiplier = 1;
    for (int p = path->size() -1; p >= 0; p--) {
        pathMapping += path->at(p) * multiplier;
        multiplier *= 100;
    }

    return pathMapping;
}

int CudaSolver::findBestMove(SlotStatus* board, Player player, int maxDepth)
{
    if (_board->IsFull()) return -1;

    auto color = SlotStatusHelpers::getSlotFromPlayer(player);
    int move = -1;
    int bestScore = INT_MIN;

    // Traverse through the board to find legal moves and see the maximum score
	// Since the board fills from the last row, it's better to traverse the 
	// board in a reverse order

    std::vector<boardAndPath> endNodes;

	for(int i = _board->getWidth() * _board->getHeight() - 1; i >= 0 ; i--) {
		if(board[i] == SlotStatus::Empty && _board->isLegalMove(i)) {
			board[i] = color;
			int score = this->minimax(board, maxDepth, player, false);
        
			board[i] = SlotStatus::Empty;
			if(score > bestScore) {
				move = i % _board->getWidth() + 1;
				bestScore = score;
			}
		}
	}

	return move;
}

int CudaSolver::findBestMove2(SlotStatus* board, Player player, int maxDepth)
{
    if (_board->IsFull()) return -1;

    auto color = SlotStatusHelpers::getSlotFromPlayer(player);
    int move = -1;
    int bestScore = INT_MIN;

    // Traverse through the board to find legal moves and see the maximum score
	// Since the board fills from the last row, it's better to traverse the 
	// board in a reverse order

    std::vector<boardAndPath> endNodes;
    std::vector<int> pathSoFar;
    findBoards2(board, _board->getWidth(), _board->getHeight(), player, maxDepth, &endNodes, pathSoFar);
    int numEndNodes = endNodes.size();
    _nodesTraversed = endNodes.size();
    int numSlots =  _board->getWidth() * _board->getHeight();
    auto concatBoards = new SlotStatus[numEndNodes * numSlots];

    std::unordered_map<uint32_t, int> pathToIndex;
    for (int n = 0; n < numEndNodes; n++) {
        for(int j = 0; j < numSlots; j++) concatBoards[n*numSlots + j] = endNodes.at(n).board[j];
        // for (auto p : endNodes.at(n).path){
        //     std::cout << p << ", ";
        // }
        // std::cout <<std::endl;
        pathToIndex[createPathMapping(&endNodes.at(n).path)] = n;
        delete[] endNodes.at(n).board;
    }

    auto scores = new double[numEndNodes];

    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    EvaluateBoards(concatBoards, numEndNodes, player, scores);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms;
    hipEventElapsedTime(&ms, start, stop);
    // Uncoment to print GPU-intensive timing part
    // std::cout << "time in GPU intense part: " << ms << std::endl;

    // std::cout << "scores ";
    // for (int as = 0; as < numEndNodes; as++) {
    //     std::cout << scores[as] << ", ";
    // }
    // std::cout <<std::endl;

	for(int i = _board->getWidth() * _board->getHeight() - 1; i >= 0 ; i--) {
		if(board[i] == SlotStatus::Empty && _board->isLegalMove(i)) {
			board[i] = color;
            std::vector<int> miniMaxPath = {i};
            auto score = minimax2(board, scores, maxDepth-1, player, false, miniMaxPath, &pathToIndex);        
    		board[i] = SlotStatus::Empty;

            // std::cout << "move " << i << " scored " << score << std::endl;

			if(score > bestScore) {
				move = i;
				bestScore = score;
			}
		}
	}

    delete[] scores;

	return move;
}

int CudaSolver::minimax(SlotStatus* board, int depth, Player player, bool maximizer)
{
    // Board evaluations are static: The player won't change, it will always be
	// the maximizer wrt whom the score will be calculated.

    if (depth == 0)	return EvaluateBoard(board, player);

	// if(score == INT_MAX || score == INT_MIN) // someone has won already
	// 	return score;

	//if(depth == 0) return score;

	auto color = SlotStatusHelpers::getSlotFromPlayer(
        (maximizer ? player : PlayerHelpers::OppositePlayer(player)));
    
    // Last stop, find all boards at this level and process in parallel
    if (depth == 1) {
        SlotStatus* concatBoard;
        int numBoards = 1;
        findBoards(board, _board->getWidth(), _board->getHeight(), color,  concatBoard, numBoards);
        auto scores  = new double[numBoards];
        int max = INT_MIN;
        int min = INT_MAX;
        EvaluateBoards(concatBoard, numBoards, maximizer ? player : PlayerHelpers::OppositePlayer(player), scores);

        for (int i = 0; i < numBoards; i++) {
            if (scores[i] > max) {
                max = scores[i];
            }
            if (scores[i] < min) {
                min = scores[i];
        }
    }

        delete[] concatBoard;
        delete[] scores;

        return maximizer ? max : min;
    }
    int bestScore = maximizer ? INT_MIN : INT_MAX;
    for(int i = _board->getWidth() * _board->getHeight() - 1; i >= 0 ; i--) {
        if(isLegalMove(board, _board->getWidth(), _board->getHeight(), i)) {
            board[i] = color;
            auto score = this->minimax(board, depth - 1, player, !maximizer);
            bestScore = maximizer ? std::max(score, bestScore) : std::min(score, bestScore);
            board[i] = SlotStatus::Empty;
            ++_nodesTraversed;
        }
    }
    
	return bestScore;
}

int CudaSolver::minimax2(SlotStatus* board, const double *scores, const int depth, const Player player, const bool maximizer, std::vector<int> pathSoFar, const std::unordered_map<uint32_t, int> *pathToIndex)
{
    // Board evaluations are static: The player won't change, it will always be
	// the maximizer wrt whom the score will be calculated.
    int score;
    auto pathMapping = createPathMapping(&pathSoFar);

    // if (depth == 0) {
    //    for (auto p : pathSoFar){
    //                 std::cout << p << ", ";
    //             }
    //             std::cout << std::endl;
    // }

    // Can only do this if the path mapping exists
    if (pathToIndex->find(pathMapping) != pathToIndex->end()) {
        int scoresIndex = pathToIndex->at(pathMapping);
        score = scores[scoresIndex];

        if(score == INT_MAX || score == INT_MIN) // someone has won already
            return score;

        if(depth == 0) return score;
    }

    if (depth == 0) {
        std::cerr << "Ahh didn't find a matching path!!!!!!" << std::endl;
        exit(911);
    }

	auto color = SlotStatusHelpers::getSlotFromPlayer(
        (maximizer ? player : PlayerHelpers::OppositePlayer(player)));
    
    int bestScore = maximizer ? INT_MIN : INT_MAX;
    for(int i = _board->getWidth() * _board->getHeight() - 1; i >= 0 ; i--) {
        if(isLegalMove(board, _board->getWidth(), _board->getHeight(), i)) { 
            std::vector<int> nextPath;
            std::copy(pathSoFar.begin(), pathSoFar.end(), back_inserter(nextPath));
            nextPath.push_back(i);
            
            board[i] = color;
            auto score = this->minimax2(board, scores, depth - 1, player, !maximizer, nextPath, pathToIndex);
            board[i] = SlotStatus::Empty;
            bestScore = maximizer ? std::max(score, bestScore) : std::min(score, bestScore);
        }
    }
    
	return bestScore;
}

void CudaSolver::printStats()
{
    printf("Stats!!");
}
